#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define M 4096
#define N 4096
#define P 4096
#define BLOCKS (M * P - 1)/1024 + 1

__global__ void matrixMultiply(int *out, int *matrixA, int *matrixB)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId < M * P)
	{
		int row = threadId % M;
		int column = threadId % P;
		int sum = 0;
		for (int i = 0; i < N; i++)
		{
			sum += *(matrixA + i + N *row) * *(matrixB + i * P + column);
		}
		*(out + column * M + row) = sum;
	}
}

int main()
{
	int *matrixA = (int*)malloc(M * N * sizeof(int));
	int *matrixB = (int*)malloc(N * P * sizeof(int));
	int *out = (int*)malloc(M * P * sizeof(int));
	int *d_matrixA, *d_matrixB, *d_out;
	int threadCount = 1024;
	if (BLOCKS == 1)
	{
		threadCount = M * P;
	}

	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < N; j++)
		{
			*(matrixA + i * N + j) = rand();
		}
	}

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < P; j++)
		{
			*(matrixB + i * P + j) = rand();
		}
	}

	//allocate device memory for matrix A
	hipMalloc((void**)&d_matrixA, M * N * sizeof(int));
	//transfer matrix A from host to device memory
	hipMemcpy(d_matrixA, matrixA, M * N * sizeof(int), hipMemcpyHostToDevice);
	//allocate device memory for matrix B
	hipMalloc((void**)&d_matrixB, N * P * sizeof(int));
	//transfer matrix B from host to device memory
	hipMemcpy(d_matrixB, matrixB, N * P * sizeof(int), hipMemcpyHostToDevice);
	//allocate device memory for output
	hipMalloc((void**)&d_out, M * P * sizeof(int));
	//Threads are N * P, but split into blocks, where appropriate
	matrixMultiply<<<BLOCKS, threadCount>>>(d_out, d_matrixA, d_matrixB);
	//transfer output from device memory to host memory
	hipMemcpy(out, d_out, M * P * sizeof(int), hipMemcpyDeviceToHost);

	//For printing the output, if necessary
	/*
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < P; j++)
		{
			printf("%d ", *(out + i + j * M));
		}
		printf("\n");
	}
	*/

	//free device memory
	hipFree(d_matrixA);
	hipFree(d_matrixB);
	hipFree(d_out);
	//free host memory
	free(matrixA);
	free(matrixB);
	free(out);
    return 0;
}