#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <vector>

#include <opencv2\opencv.hpp>
#include <opencv2\core\core_c.h>
#include <opencv2\highgui\highgui_c.h>

using namespace cv;
using namespace std;

class camera {
	//Focal point
	float focalP[3];
	//Focal length
	float focalL;
	int window[2] = { 1 , 0 };
	float up[3];
	float right[3];
	float forwards[3];
	float topLeft[3];
	float pixelSize[2];
public:
	camera(float c_focalP[], float c_focalL, int x, int y, float c_up[], float c_right[], float c_forwards[]) {
		for (int i = 0; i < 3; i++) {
			focalP[i] = c_focalP[i];
		}
		focalL = c_focalL;
		window[1] = x / y;
		for (int i = 0; i < 3; i++) {
			up[i] = c_up[i];
		}
		for (int i = 0; i < 3; i++) {
			right[i] = c_right[i];
		}
		for (int i = 0; i < 3; i++) {
			forwards[i] = c_forwards[i];
		}
		pixelSize[0] = window[1] / y;
		pixelSize[1] = window[2] / x;
		for (int i = 0; i < 3; i++) {
			topLeft[i] = focalP[i] + focalL*forwards[i] + window[1]*up[i] - window[2]*right[i] - pixelSize[1]*up[i] + pixelSize[2]*right[i];
		}
	}
};

class light {
	float direction[3];
	float ambientIntensity;
	float localIntensity;
public:
	light(float l_direction[], float l_ambient, float l_local) {
		float norm = sqrtf(l_direction[0] * l_direction[0] + l_direction[1] * l_direction[1] + l_direction[2] * l_direction[2]);
		for (int i = 0; i < 3; i++){
			direction[i] = l_direction[i] / norm;
		}
		ambientIntensity = l_ambient;
		localIntensity = l_local;
	}
};

//custom meshes
class mesh {

};

class sphere {
	float radius;
	float centre[3];
	float colour[3];
	float diffuseInt;
	float diffuse[3];
	float specularInt;
	float specular;
	float ambientInt;
public:
	sphere(float s_radius, float s_centre[3], float s_colour[3], float s_diffuseInt, float s_specularInt, float specular) {
		radius = s_radius;
		for (int i = 0; i < 3; i++) {
			centre[i] = s_centre[i];
		}
		for (int i = 0; i < 3; i++) {
			colour[i] = s_colour[i];
		}
		diffuseInt = s_diffuseInt;
		for (int i = 0; i < 3; i++) {
			diffuse[i] = s_colour[i];
		}
		specularInt = s_specularInt;
		specular = specular;
		ambientInt = 1 - specularInt - diffuseInt;
	}
};

__device__ int* getColour()
{

}

__global__ void getPixel(float *out, camera *cam, light *light, int *x, int *y)
{
	int c = blockIdx.x*blockDim.x + threadIdx.x;
	int r = blockIdx.y*blockDim.y + threadIdx.y;
	if ((c < *x) && (r < *y))
	{
		for (int i = 0; i < 3; i++)
		{
			float PLACEHOLDER = 128;
			*(out + c * *y * 3 + r * 3 + i) = PLACEHOLDER;
		}
	}
	//__syncthreads();
}

void makeImage(Mat &mat, float *image, int *y)
{
	CV_Assert(mat.channels() == 4);
	for (int i = 0; i < mat.rows; ++i) {
		for (int j = 0; j < mat.cols; ++j) {
			Vec4b& bgra = mat.at<Vec4b>(i, j);
			bgra[0] = saturate_cast<uchar>(*(image + i * *y * 3 + j * 3)); //Blue
			bgra[1] = saturate_cast<uchar>(*(image + i * *y * 3 + j * 3 + 1)); //Green
			bgra[2] = saturate_cast<uchar>(*(image + i * *y * 3 + j * 3 + 2)); //Red
			bgra[3] = saturate_cast<uchar>((float)255.0); //Alpha
		}
	}
}

int main()
{
	int *x = (int*)malloc(sizeof(int));
	*x = 1920;
	int *y = (int*)malloc(sizeof(int));
	*y = 1080;
	float focalP[] = {0, 0, 0};
	float focalL = 1;
	float up[3] = { 0, 1, 0 };
	float right[3] = { 1, 0, 0 };
	float forwards[] = { 0, 0, 1 };
	camera *camera1 = (camera*)malloc(sizeof(camera));
	//camera
	*camera1 = camera(focalP, focalL, *x, *y, up, right, forwards);
	float lightDirection[] = { -2, 1, -3 };
	float ambientInt = 0.2;
	float localInt = 0.8;
	light *light1 = (light*)malloc(sizeof(light));
	//light
	*light1 = light(lightDirection, ambientInt, localInt);
	//limited 1024
	dim3 block(32, 32, 1);
	dim3 grid;
	grid.x = (*x + block.x - 1) / block.x;
	grid.y = (*y + block.y - 1) / block.y;
	float *out = (float*)malloc(*x * *y * 3 * sizeof(float));
	for (int i = 0; i < *x; i++)
	{
		for (int j = 0; j < *y; j++)
		{
			for (int k = 0; k < 3; k++)
			{
				*(out + i * *y * 3 + j * 3 + k) = (float)51;
			}
		}
	}
	int *d_x, *d_y;
	float *d_out;
	camera *d_camera1;
	light *d_light1;
	//allocate device memory for variables
	hipMalloc((void**)&d_x, sizeof(int));
	hipMalloc((void**)&d_y, sizeof(int));
	hipMalloc((void**)&d_out, *x * *y * 3 * sizeof(float));
	hipMalloc((void**)&d_camera1, sizeof(camera));
	hipMalloc((void**)&d_light1, sizeof(light));
	//transfer from host to device memory
	hipMemcpy(d_x, x, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_camera1, camera1, sizeof(camera), hipMemcpyHostToDevice);
	hipMemcpy(d_light1, light1, sizeof(light), hipMemcpyHostToDevice);
	getPixel<<<grid, block>>>(d_out, d_camera1, d_light1, d_x, d_y);
	//transfer output from device memory to host memory
	hipMemcpy(out, d_out, *x * *y * 3 * sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	//free device memory
	hipFree(d_camera1);
	hipFree(d_light1);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_out);
	//Opencv documentation approach to saving an image
	Mat mat(*y, *x, CV_8UC4);
	makeImage(mat, out, y);
	vector<int> compression_params;
	compression_params.push_back(IMWRITE_PNG_COMPRESSION);
	compression_params.push_back(9);
	try {
		imwrite("output.png", mat, compression_params);
	}
	catch (runtime_error& ex) {
		fprintf(stderr, "Exception converting image to PNG format: %s\n", ex.what());
		return 1;
	}
	fprintf(stdout, "Saved PNG file with alpha data.\n");
	//free host memory
	free(x);
	free(y);
	free(camera1);
	free(light1);
	free(out);
	return 0;
}