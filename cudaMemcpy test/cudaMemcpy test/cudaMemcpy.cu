
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void add(int *c, const int *a, const int *b)
{
	*c = *a + *b;
}

int main()
{
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	//allocate space for device copies of a, b, c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	a = 2;
	b = 7;

	//copy  inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<1, 1>>>(d_c, d_a, d_b);

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("%d\n", c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
