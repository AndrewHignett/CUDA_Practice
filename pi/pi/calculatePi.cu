#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>

#define N 1024
#define blockCount 1

__global__ void calculatePi(float *out)
{
	//Shared memeory for sum, only works with a blockCount of 1
	__shared__ float cache[N];
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	int stepSize = blockDim.x * gridDim.x;

	// y = root(1 - x^2)
	// calculate y from x = threadID/N (giving an over estimate)
	//sum of all y*x*4 = solution
	
	float x = float(1) /(blockCount*N);
	float thisX = float(threadID) / (blockCount*N);
	float y = sqrt(1 - thisX * thisX);
	float area = y * x * 4;
	printf("ThreadID: %d\nthisX: %f\nX: %f\nY: %f\nArea: %f\n", threadID, thisX, x, y, area);
	cache[threadID] = area;
	__syncthreads();
	
	if (threadIdx.x == 0)
	{
		float sum = 0.0;
		for (int i = 0; i < N; i++)
		{
			sum += cache[i];
		}
		*out = sum;
	}
}

int main()
{
	float out, *d_out;
	hipMalloc((void**)&d_out, sizeof(float));
	hipMemcpy(d_out, &out, sizeof(float), hipMemcpyHostToDevice);
	//Parallel pi calculation, single block
	calculatePi<<<blockCount, N >>>(d_out);
	hipMemcpy(&out, d_out, sizeof(float), hipMemcpyDeviceToHost);
	printf("%f\n", out);
	hipFree(d_out);
	return 0;
}