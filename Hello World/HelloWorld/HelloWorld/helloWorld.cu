#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void helloWorld() {
	printf("Hello World\n");
}

int main() {
	helloWorld <<<1, 1>>> ();
	return 0;
}